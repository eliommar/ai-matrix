
#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand.h>
#include <hipblas.h>
#include <iomanip>

#define MAX(x, y) ((x>y) ? x : y)
// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
	if (stat != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
	}
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
	}
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
	if (stat != HIPRAND_STATUS_SUCCESS) {
		fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
	}
}


double cal_tflops(int m, int n, int k, double msec)
{
    double flops = 2. * m * n * k;
    double tflops = (1E-12*flops) / (1E-3*msec);
    return tflops;
}

 

__global__ void assignFloatValue (float *out, int n, float value) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < n) {
		out[idx] = value;
	}
}

__global__ void assignHalfValue (half *out, int n, float value) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < n) {
		out[idx] = value;
	}
}
void correctnessCheck(int m, int n, int k, float *host, float value){
        for (int i = 0; i < m * n; i++) {      
            float val = host[i];
            if ( val != k * value * value) {
                std::cout << "ERROR value = " << val<< std::endl;
            }
        }
}

void printTime(float cublasTime, int m, int n, int k, float &s_max_tflops){
        float tflops = cal_tflops(m, n, k, cublasTime);
        s_max_tflops = MAX(tflops, s_max_tflops);
        std::cout << std::setw(7) << m << ",";
        std::cout << std::setw(7) << n << ",";
        std::cout << std::setw(7) << k << ",";
        std::cout << std::setw(15) << std::setprecision(4) << cublasTime << ",";
        std::cout << std::setw(15) << std::setprecision(4) << tflops << "," << std::endl;
}


int main(int argc, char* argv[]) {
    int m,n,k;
    int start = 512;
    int end = 10240;
    if (argc < 3) {
        return 0;
    }
    start = std::atoi(argv[1]);
    end = std::atoi(argv[2]);
    
    
    std::cout << "[TensorCore INT8(INT32 accumulation) Time and TOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TOPS";
    std::cout << std::endl;
    float s_max_tflops = 0;
    // for tensorcore test
    for (int i=start; i<=end; i+= 1024){
        m = n = k = i;
  
        int8_t *a_fp16;
        int8_t *b_fp16;
        int *c_cublas;
        int *c_host_cublas;
        //const int  value = 1;

   
        hipblasHandle_t cublasHandle;

        hipEvent_t startcublas;
        hipEvent_t stopcublas;

        cudaErrCheck(hipEventCreate(&startcublas));
        cudaErrCheck(hipEventCreate(&stopcublas));
        cublasErrCheck(hipblasCreate(&cublasHandle));
        // Use tensor cores
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

        cudaErrCheck(hipMalloc((void**)&a_fp16, m * k * sizeof(int8_t)));
        cudaErrCheck(hipMalloc((void**)&b_fp16, k * m * sizeof(int8_t)));
        cudaErrCheck(hipMalloc((void**)&c_cublas, m * n * sizeof(int)));
        c_host_cublas = (int*)malloc(m * n * sizeof(int));

        //TODO curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
        //assignHalfValue <<< (m * k + 255) / 256, 256 >>> (a_fp16, m*k, value);
        //assignHalfValue <<< (k * n + 255) / 256, 256 >>> (b_fp16, k*n, value);
        //assignHalfValue <<< (k * n + 255) / 256, 256 >>> (c_cublas, m*n, 0.0f);

        int alpha = 1;
        int beta = 0;
        int numRepeats = 50;
        // Warp up
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
            m, n, k, 
            &alpha,
            a_fp16, HIP_R_8I, m,
            b_fp16, HIP_R_8I, k,
            &beta, 
            c_cublas, HIP_R_32I, m,
            HIP_R_32I, CUBLAS_GEMM_DFALT_TENSOR_OP));

        // Now using cuBLAS
        cudaErrCheck(hipEventRecord(startcublas));
        for (int iteration = 0; iteration < numRepeats; ++iteration) {
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    m, n, k, 
                    &alpha,
                    a_fp16, HIP_R_8I, m,
                    b_fp16, HIP_R_8I, k,
                    &beta, 
                    c_cublas, HIP_R_32I, m,
                    HIP_R_32I, CUBLAS_GEMM_DFALT_TENSOR_OP));
        }
        cudaErrCheck(hipEventRecord(stopcublas));
        cudaErrCheck(hipEventSynchronize(stopcublas));
        // TODO: Correctness check
        //cudaErrCheck(cudaMemcpy(c_host_cublas, c_cublas, m * n * sizeof(float), cudaMemcpyDeviceToHost));
       //correctnessCheck(m, n, k, c_host_cublas, value);
        // Check time
        float cublasTime;	
        cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas)); 
        cublasTime /= numRepeats;
        printTime(cublasTime, m, n, k, s_max_tflops);
        
        cudaErrCheck(hipEventDestroy(startcublas));             
        cudaErrCheck(hipEventDestroy(stopcublas));
        cudaErrCheck(hipFree(a_fp16));
        cudaErrCheck(hipFree(b_fp16));
        cudaErrCheck(hipFree(c_cublas));
        free(c_host_cublas);
    }
    std::cout << "[Peak TOPS]" << std::endl << std::setprecision(4) << s_max_tflops << std::endl;
    cudaErrCheck(hipDeviceReset());
    

    std::cout << "[TensorCore FP16(FP16 accumulation) Time and TFLOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TFLOPS";
    std::cout << std::endl;
    s_max_tflops = 0;
    // for tensorcore test
    for (int i=start; i<=end; i+= 1024){
        m = n = k = i;
  
        half *a_fp16;
        half *b_fp16;
        half *c_cublas;
        float *c_host_cublas;
        const float  value = 1.0f;

   
        hipblasHandle_t cublasHandle;

        hipEvent_t startcublas;
        hipEvent_t stopcublas;

        cudaErrCheck(hipEventCreate(&startcublas));
        cudaErrCheck(hipEventCreate(&stopcublas));
        cublasErrCheck(hipblasCreate(&cublasHandle));
        // Use tensor cores
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

        cudaErrCheck(hipMalloc((void**)&a_fp16, m * k * sizeof(half)));
        cudaErrCheck(hipMalloc((void**)&b_fp16, k * m * sizeof(half)));
        cudaErrCheck(hipMalloc((void**)&c_cublas, m * n * sizeof(half)));
        c_host_cublas = (float*)malloc(m * n * sizeof(float));

        // curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
        assignHalfValue <<< (m * k + 255) / 256, 256 >>> (a_fp16, m*k, value);
        assignHalfValue <<< (k * n + 255) / 256, 256 >>> (b_fp16, k*n, value);
        assignHalfValue <<< (k * n + 255) / 256, 256 >>> (c_cublas, m*n, 0.0f);

        float alpha = 1.0f;
        float beta = 0.0f;
        int numRepeats = 50;
        // Warp up
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
            m, n, k, 
            &alpha,
            a_fp16, HIP_R_16F, m,
            b_fp16, HIP_R_16F, k,
            &beta, 
            c_cublas, HIP_R_16F, m,
            HIP_R_16F, CUBLAS_GEMM_DFALT_TENSOR_OP));

        // Now using cuBLAS
        cudaErrCheck(hipEventRecord(startcublas));
        for (int iteration = 0; iteration < numRepeats; ++iteration) {
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    m, n, k, 
                    &alpha,
                    a_fp16, HIP_R_16F, m,
                    b_fp16, HIP_R_16F, k,
                    &beta, 
                    c_cublas, HIP_R_16F, m,
                    HIP_R_16F, CUBLAS_GEMM_DFALT_TENSOR_OP));
        }
        cudaErrCheck(hipEventRecord(stopcublas));
        cudaErrCheck(hipEventSynchronize(stopcublas));
        // TODO: Correctness check
        //cudaErrCheck(cudaMemcpy(c_host_cublas, c_cublas, m * n * sizeof(float), cudaMemcpyDeviceToHost));
       //correctnessCheck(m, n, k, c_host_cublas, value);
        // Check time
        float cublasTime;	
        cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas)); 
        cublasTime /= numRepeats;
        printTime(cublasTime, m, n, k, s_max_tflops);
        
        cudaErrCheck(hipEventDestroy(startcublas));             
        cudaErrCheck(hipEventDestroy(stopcublas));
        cudaErrCheck(hipFree(a_fp16));
        cudaErrCheck(hipFree(b_fp16));
        cudaErrCheck(hipFree(c_cublas));
        free(c_host_cublas);
    }
    std::cout << "[Peak TFLOPS]" << std::endl << std::setprecision(4) << s_max_tflops << std::endl;
    cudaErrCheck(hipDeviceReset());
    
 
    std::cout << "[TensorCore FP16(FP32 accumulation) Time and TFLOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TFLOPS";
    std::cout << std::endl;
    s_max_tflops = 0;
    // for tensorcore test
    for (int i=start; i<=end; i+= 1024){
        m = n = k = i;
  
        half *a_fp16;
        half *b_fp16;
        float *c_cublas;
        float *c_host_cublas;
        const float  value = 1.0f;

   
        hipblasHandle_t cublasHandle;

        hipEvent_t startcublas;
        hipEvent_t stopcublas;

        cudaErrCheck(hipEventCreate(&startcublas));
        cudaErrCheck(hipEventCreate(&stopcublas));
        cublasErrCheck(hipblasCreate(&cublasHandle));
        // Use tensor cores
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

        cudaErrCheck(hipMalloc((void**)&a_fp16, m * k * sizeof(half)));
        cudaErrCheck(hipMalloc((void**)&b_fp16, k * m * sizeof(half)));
        cudaErrCheck(hipMalloc((void**)&c_cublas, m * n * sizeof(float)));
        c_host_cublas = (float*)malloc(m * n * sizeof(float));

        // curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
        assignHalfValue <<< (m * k + 255) / 256, 256 >>> (a_fp16, m*k, value);
        assignHalfValue <<< (k * n + 255) / 256, 256 >>> (b_fp16, k*n, value);
        assignFloatValue <<< (k * n + 255) / 256, 256 >>> (c_cublas, m*n, 0.0f);

        float alpha = 1.0f;
        float beta = 0.0f;
        int numRepeats = 50;
        // Warp up
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
            m, n, k, 
            &alpha,
            a_fp16, HIP_R_16F, m,
            b_fp16, HIP_R_16F, k,
            &beta, 
            c_cublas, HIP_R_32F, m,
            HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP));

        // Now using cuBLAS
        cudaErrCheck(hipEventRecord(startcublas));
        for (int iteration = 0; iteration < numRepeats; ++iteration) {
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    m, n, k, 
                    &alpha,
                    a_fp16, HIP_R_16F, m,
                    b_fp16, HIP_R_16F, k,
                    &beta, 
                    c_cublas, HIP_R_32F, m,
                    HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP));
        }
        cudaErrCheck(hipEventRecord(stopcublas));
        cudaErrCheck(hipEventSynchronize(stopcublas));
        // Correctness check
        cudaErrCheck(hipMemcpy(c_host_cublas, c_cublas, m * n * sizeof(float), hipMemcpyDeviceToHost));
        correctnessCheck(m, n, k, c_host_cublas, value);
        // Check time
        float cublasTime;	
        cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas)); 
        cublasTime /= numRepeats;
        printTime(cublasTime, m, n, k, s_max_tflops);
        
        cudaErrCheck(hipEventDestroy(startcublas));             
        cudaErrCheck(hipEventDestroy(stopcublas));
        cudaErrCheck(hipFree(a_fp16));
        cudaErrCheck(hipFree(b_fp16));
        cudaErrCheck(hipFree(c_cublas));
        free(c_host_cublas);
    }
    std::cout << "[Peak TFLOPS]" << std::endl << std::setprecision(4) << s_max_tflops << std::endl;
    cudaErrCheck(hipDeviceReset());

    std::cout << "[FP32 Time and TFLOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TFLOPS";
    std::cout << std::endl;
    s_max_tflops = 0;
    // for float test
    for (int i=start; i<=end; i+= 1024){
        m = n = k = i;
  
        float *a_fp32;
        float *b_fp32;
        float *c_cublas;
        float *c_host_cublas;
        const float  value = 1.0f;

   
        hipblasHandle_t cublasHandle;

        hipEvent_t startcublas;
        hipEvent_t stopcublas;

        cudaErrCheck(hipEventCreate(&startcublas));
        cudaErrCheck(hipEventCreate(&stopcublas));
        cublasErrCheck(hipblasCreate(&cublasHandle));
        // No tensor cores
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_DEFAULT_MATH));

        cudaErrCheck(hipMalloc((void**)&a_fp32, m * k * sizeof(float)));
        cudaErrCheck(hipMalloc((void**)&b_fp32, k * m * sizeof(float)));
        cudaErrCheck(hipMalloc((void**)&c_cublas, m * n * sizeof(float)));
        c_host_cublas = (float*)malloc(m * n * sizeof(float));

        // curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
        assignFloatValue <<< (m * k + 255) / 256, 256 >>> (a_fp32, m*k, value);
        assignFloatValue <<< (k * n + 255) / 256, 256 >>> (b_fp32, k*n, value);
        assignFloatValue <<< (k * n + 255) / 256, 256 >>> (c_cublas, m*n, 0.0f);

        float alpha = 1.0f;
        float beta = 0.0f;
        int numRepeats = 50;
        // warp up
        cublasErrCheck(hipblasSgemm(cublasHandle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_N,
                m,
                n,
                k,
                &alpha,
                a_fp32, m,
                b_fp32, k,
                &beta,
                c_cublas, m)); 
        
        cudaErrCheck(hipEventRecord(startcublas));
        for (int iteration = 0; iteration < numRepeats; ++iteration) {
            cublasErrCheck(hipblasSgemm(cublasHandle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_N,
                m,
                n,
                k,
                &alpha,
                a_fp32, m,
                b_fp32, k,
                &beta,
                c_cublas, m)); 
        }
        cudaErrCheck(hipEventRecord(stopcublas));
        cudaErrCheck(hipEventSynchronize(stopcublas));
        // Correctness check
        cudaErrCheck(hipMemcpy(c_host_cublas, c_cublas, m * n * sizeof(float), hipMemcpyDeviceToHost));
        correctnessCheck(m, n, k, c_host_cublas, value);
        // Check time
        float cublasTime = 0.0f;	
        cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas)); 
        cublasTime /= numRepeats;
        printTime(cublasTime, m, n, k, s_max_tflops);
        
        cudaErrCheck(hipEventDestroy(startcublas));             
        cudaErrCheck(hipEventDestroy(stopcublas));
        cudaErrCheck(hipFree(a_fp32));
        cudaErrCheck(hipFree(b_fp32));
        cudaErrCheck(hipFree(c_cublas));
        free(c_host_cublas);
    }
    std::cout << "[Peak TFLOPS]" << std::endl << std::setprecision(4) << s_max_tflops << std::endl;
    cudaErrCheck(hipDeviceReset());
    


	return 0;
}
